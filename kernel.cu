﻿
#include "Matrix.cuh"

using T = unsigned long;
float Matrix<T>::last_multiplication_time = 0.0f;

/*
	filename[0] Matrix A file
	filename[1] Matrix B file
	filename[2] Output file
*/
bool check_python(const std::string filename[3])
{
	const char* verificaton_result = "verification.txt";

	std::cout << "Checking results by Python's NumPy";
	system(("python verificatior.py " + filename[0] + ' ' + filename[1] + ' ' + filename[2] + " > " + verificaton_result).c_str());

	std::ifstream file;
	file.exceptions(std::ifstream::badbit);
	file.open(verificaton_result);

	std::string buffer = "False";
	getline(file, buffer);
	file.close();
	remove(verificaton_result);

	return buffer != "False";
}

int main(int argc, char** argv)
{
	system("title Parallel Programming [Lab №4]");
	if (argc != 1 && argc != 4 && (argc == 2 && strcmp(argv[1], "--help") == 0))
	{
		std::cout << "Locate paths to matrix files in arguments, to output file and specify number of threads\n\n"
			<< "EXAMPLE:\n"
			<< "    .../PP_3.exe <matrix_1_path> <matrix_2_path> <output_path>\n\n";
		exit(EXIT_SUCCESS);
	}

	/*
	[0] Matrix A file
	[1] Matrix B file
	[2] Output file
	*/
	std::string filename[3];

	if (argc == 4)
	{
		for (int i = 1; i < argc; i++)
			filename[i] = argv[i];
	}
	else
	{
		std::cout << "Locate path to matrix A: ";
		std::cin >> filename[0];

		std::cout << "Locate path to matrix B: ";
		std::cin >> filename[1];

		std::cout << "Locate path to output file: ";
		std::cin >> filename[2];
		std::cout << '\n';
	}

	try
	{

		std::cout << "Reading matrix A";
		Matrix<T> a(filename[0]);

		std::cout << "\rReading matrix B";
		Matrix<T> b(filename[1]);

		std::cout << "\rPerforming C = A * B";
		Matrix<T> c = a * b;

		std::cout << "\rWriting matrix C to file [" << filename[2] << "]";
		c.write_file(filename[2]);
		std::cout << '\r' << std::string(filename[2].size() + 40, ' ') << '\r';

		if (check_python(filename))
		{
			std::cout << "\rAdding multiplication results in [" << filename[2] << "]...";
			c.write_multiplication_result(filename[2]);
			c.write_multiplication_result("res/cuda_res.txt");

			std::cout << '\r' << std::string(filename[2].size() + 40, ' ');

			std::cout << "\rMatrix multiplication was done correctly\n"
				"See results in [" << filename[2] << "]";
		}
		else
			std::cout << "\rMatrix multiplication wasn't done correctly";
	}
	catch (std::exception const& ex)
	{
		std::cout << "\n\n[!] ERROR [!]\n" << ex.what() << "\n\n";
		exit(EXIT_FAILURE);
	}

	std::cout << "\n\n";
	return 0;
}
